#include "hip/hip_runtime.h"
#include "kernelFunctions.h"
#include "globalsingleton.h"

void checkCUDAError(const char *msg, int syncFlag) {
  if (syncFlag)
    // block until the device has completed                                                                                                    
    hipDeviceSynchronize();

  hipError_t error = hipGetLastError();

  if (hipSuccess != error) {
    fprintf(stderr,"CUDA ERROR: %s: %s\n", msg, hipGetErrorString(error));
    exit(1);
  }
}



/**
 *  A convenience function to be used on the GPU device to validate
 *  requested vector indices.
 **/
__device__ int gpuIsValidVectorIdx(const int idx, const int n) {
  return ( (idx >= 0) && (idx < n) );
}



//-----------------------------------------------------------------------------------------

/**
 *  Perform the GPU-based operation y=ax+y, where x and y are vectors
 *  and a is a scalar.  The routine uses only the GPU-based memory
 *  for vector values and so assumes the caller will copy CPU data
 *  into the GPU before calling this and copy results back when
 *  they are ready to do so.
 **/
__global__ void gpuSumAXplusThisKernel(const int n, 
				    const REAL_TYPE a, 
				    const REAL_TYPE *x, 
				    REAL_TYPE *y) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x; 
    if (gpuIsValidVectorIdx(idx,n))
      y[idx] += a*x[idx];
}

//extern "C" 
void launch_SumAXplusThisKernel(int BLOCK_N, int THREAD_N, const hipStream_t &s, 
				const int n, const REAL_TYPE a, const REAL_TYPE *x, REAL_TYPE *y) {
  gpuSumAXplusThisKernel<<<BLOCK_N, THREAD_N, 0, s>>>(n,a,x,y);
}




//-----------------------------------------------------------------------------------------

/**
 *  Perform the GPU-based operation z=x+y, where x, y, and z are
 *  vectors.  The routine uses only the GPU-based memory for vector
 *  values and so assumes the caller will copy CPU data into the GPU
 *  before calling this and copy results back when they are ready to
 *  do so.
 **/
__global__ void gpuSumXplusYKernel(const int n, 
				   const REAL_TYPE *x, 
				   const REAL_TYPE *y, 
				   REAL_TYPE *z) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x; 
  if (gpuIsValidVectorIdx(idx,n))
     z[idx] = x[idx] + y[idx];
}

//extern "C" 
void launch_SumXplusYKernel(int BLOCK_N, int THREAD_N, const hipStream_t &s, 
			    const int n, const REAL_TYPE *x, const REAL_TYPE *y, REAL_TYPE *z) {
  gpuSumXplusYKernel<<<BLOCK_N, THREAD_N, 0, s>>>(n,x,y,z);
}




//-----------------------------------------------------------------------------------------

/**
 *  Perform the GPU-based operation y=x, where x and y are
 *  vectors.  The routine uses only the GPU-based memory for vector
 *  values and so assumes the caller will copy CPU data into the GPU
 *  before calling this and copy results back when they are ready to
 *  do so.
 **/
__global__ void gpuYeqXKernel(const int n, 
			      const REAL_TYPE *x, 
			      REAL_TYPE *y) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x; 
  if (gpuIsValidVectorIdx(idx,n))
     y[idx] = x[idx];
}

//extern "C" 
void launch_YeqXKernel(int BLOCK_N, int THREAD_N, const hipStream_t &s, 
			    const int n, const REAL_TYPE *x, REAL_TYPE *y) {
  gpuYeqXKernel<<<BLOCK_N, THREAD_N, 0, s>>>(n,x,y);
}



//-----------------------------------------------------------------------------------------

/**
 * Sum all values in an array on the GPU.  CAUTION:  This assumes only
 * one thread of one block is being used.  It's here simply as a convenience
 * for the GPU-based dot product routine to avoid copying the accumulated 
 * results back into CPU memory.
 **/
__global__ void gpuAccumulateKernel(const int n, const REAL_TYPE *x, REAL_TYPE *total) {
  total[0]=0.0;
  for (int idx=0.0; idx<n; idx++)
    total[0] += x[idx];
}

//extern "C" 
void launch_AccumulateKernel(int BLOCK_N, int THREAD_N, const hipStream_t &s, 
			    const int n, const REAL_TYPE *x, REAL_TYPE *total) {
  gpuAccumulateKernel<<<BLOCK_N, THREAD_N, 0, s>>>(n,x,total);
}



/**
 * Perform most of a dot product operation on the GPU using only GPU-based
 * memory structures.  This routine will result in a reduced set of summed
 * values in z, which is merely an accumulator array of length equal to
 * the number of blocks in the grid.
 **/
__global__ void gpuDotProductKernel(const int n, const REAL_TYPE *x, const REAL_TYPE *y, REAL_TYPE *z ) { 
  // Shared buffer for accumulating individual thread results
  __shared__ REAL_TYPE cache[1024];//GlobalSingleton::getThreadsPerBlock()]; 

  int idx = threadIdx.x + blockIdx.x * blockDim.x; 
  int cacheIndex = threadIdx.x;
  REAL_TYPE temp = 0; 

  while (idx < n) {
    temp += x[idx] * y[idx]; 
    idx += blockDim.x * gridDim.x;
  }

  // set the cache values
  cache[cacheIndex] = temp;

  // Make sure all threads have completed their accumulation
  // from their respective blocks before proceeding (i.e.,
  // the cache is filled)
  __syncthreads();

  // For reductions, threadsPerBlock must be a power of 2 
  // because of the following code 
  int i = blockDim.x/2; 
  while (i != 0) {
    if (cacheIndex < i) 
      cache[cacheIndex] += cache[cacheIndex + i];
    __syncthreads(); 
    i /= 2;
  }

  if (cacheIndex == 0) 
    z[blockIdx.x] = cache[0];
}

//extern "C" 
void launch_DotProductKernel(int BLOCK_N, int THREAD_N, const hipStream_t &s, 
			     const int n, const REAL_TYPE *x, const REAL_TYPE *y, REAL_TYPE *z) {
  gpuDotProductKernel<<<BLOCK_N, THREAD_N>>>(n,x,y,z);
}




//-----------------------------------------------------------------------------------------

/**
 * This is a GPU-based operation to scale a vector (in-place):  x = ax
 **/
__global__ void gpuScaleVectorKernel(const int n,
				   REAL_TYPE *x, 
				   const REAL_TYPE a) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x; 
  if (gpuIsValidVectorIdx(idx,n))
     x[idx] = a * x[idx] ;
}

void launch_ScaleVectorKernel(int BLOCK_N, int THREAD_N, const hipStream_t &s, 
			    const int n, REAL_TYPE *x, const REAL_TYPE a) {
  gpuScaleVectorKernel<<<BLOCK_N, THREAD_N, 0, s>>>(n,x,a);
}



//-----------------------------------------------------------------------------------------

/**
 * The is a GPU-based operation to produce a new vector by scaling
 * another vector:  y = ax
 **/
__global__ void gpuEqualsScaleVectorKernel(const int n,
					   REAL_TYPE *y, 
					   const REAL_TYPE a,
					   const REAL_TYPE *x) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x; 
  if (gpuIsValidVectorIdx(idx,n))
     y[idx] = a * x[idx] ;
}

void launch_EqualsScaleVectorKernel(int BLOCK_N, int THREAD_N, const hipStream_t &s, 
				    const int n, REAL_TYPE *y, const REAL_TYPE a, const REAL_TYPE *x) {
  gpuEqualsScaleVectorKernel<<<BLOCK_N, THREAD_N, 0, s>>>(n,y,a,x);
}




//-----------------------------------------------------------------------------------------

/**
 * This is a GPU-based operation to produce a new vector by adding two other
 * vectors:  z = x + y
 **/
__global__ void gpuEqualsXplusYKernel(const int n,
				      REAL_TYPE *z, 
				      const REAL_TYPE *x, 
				      const REAL_TYPE *y) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x; 
  if (gpuIsValidVectorIdx(idx,n))
     z[idx] = x[idx] + y[idx];
}

void launch_EqualsXplusYKernel(int BLOCK_N, int THREAD_N, const hipStream_t &s, 
			       const int n, REAL_TYPE *z, const REAL_TYPE *x, const REAL_TYPE *y) {
  gpuEqualsXplusYKernel<<<BLOCK_N, THREAD_N, 0, s>>>(n,z,x,y);
}




//-----------------------------------------------------------------------------------------

/**
 * Use the GPU to perform the operation Ax=y, where A is a matrix,
 * and x and y are vectors.  This routine uses only the on-board
 * GPU memory and expects that data from the CPU storage has already
 * been copied.  It DOES NOT copy the GPU-stored results structure
 * back to the CPU memory.
 **/
__global__ void gpuMultMatAbyVecXKernel(int numRows, 
					int numCols, 
					int numColsPerRow,
					int whichDevice,
					int vectorOffsetPos,
                                        int numComp,
					const int *indices, 
					const REAL_TYPE *data,
					REAL_TYPE *x, 
					REAL_TYPE *y) {
  int row = blockDim.x * blockIdx.x + threadIdx.x;
  int compRow = row + vectorOffsetPos;	
 
  if (row < numComp) {
    y[compRow] = 0.0;

    for (int idx=0; idx<numColsPerRow; idx++) {
      int pos = row*numColsPerRow + idx;
      int col = indices[pos];
      REAL_TYPE value = data[pos];

      if (col != INVALID_COLUMN)
        y[compRow] += value * x[col];
    }
  }
}

extern "C" 
void launch_MultMatAbyVecXKernel(int BLOCK_N, int THREAD_N, const hipStream_t &s, 
				 int numRows, int numCols, int numColsPerRow,
				 int whichDevice, int vectorOffsetPos, int numComp, const int *indices, 
				 const REAL_TYPE *data, REAL_TYPE *x, REAL_TYPE *y) {
  gpuMultMatAbyVecXKernel<<<BLOCK_N, THREAD_N, 0, s>>>(numRows,
                                                       numCols, 
						       numColsPerRow,
						       whichDevice,
						       vectorOffsetPos,
                                                       numComp,
						       indices, data,
						       x, y);
}
